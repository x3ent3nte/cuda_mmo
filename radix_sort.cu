
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <random>

unsigned int* vals_buffer;
unsigned int* pos_buffer;

unsigned int* flags;
unsigned int* addresses;
unsigned int* block_offsets;

__device__
unsigned int floatFlip(unsigned int value)
{
	unsigned int mask = (-(value >> 31)) | 0x80000000;
	return value ^ mask;
}

__device__ 
unsigned int floatFlipInverse(unsigned int value)
{
	int mask = ((value >> 31) - 1) | 0x80000000;
	return value ^ mask;
}

__global__
void convertFloats(unsigned int* vals, unsigned int num_elems)
{
	int local_index = threadIdx.x;
	int global_index = local_index + (blockDim.x * blockIdx.x);

	if(global_index >= num_elems)
	{
		return;
	}

	vals[global_index] = floatFlip(vals[global_index]);
}

__global__
void invertFloats(unsigned int* vals, unsigned int num_elems)
{
	int local_index = threadIdx.x;
	int global_index = local_index + (blockDim.x * blockIdx.x);

	if(global_index >= num_elems)
	{
		return;
	}

	vals[global_index] = floatFlipInverse(vals[global_index]);
}

__global__
void flipFlags(unsigned int* flags, unsigned int num_elems)
{
	int local_index = threadIdx.x;
	int global_index = local_index + (blockDim.x * blockIdx.x);

	if(global_index >= num_elems)
	{
		return;
	}	

	flags[global_index] ^= 1;
}

__global__ 
void markFlags(unsigned int* vals, unsigned int* flags, bool high, unsigned int bit_pos, unsigned int num_elems)
{
	int local_index = threadIdx.x;
	int global_index = local_index + (blockDim.x * blockIdx.x);

	if(global_index >= num_elems)
	{
		return;
	}

	unsigned int value = vals[global_index]; 
	unsigned int mask = 1 << bit_pos;
	unsigned int anded = value & mask;

	if(high)
	{
		if(anded == 0)
		{
			flags[global_index] = 0;
		}
		else
		{
			flags[global_index] = 1;
		}
	}
	else
	{
		if(anded == 0)
		{
			flags[global_index] = 1;
		}
		else
		{
			flags[global_index] = 0;
		}
	}
}

__global__
void scanExclusiveSumWithBlockOffsets(unsigned int* nums, unsigned int* c_nums, unsigned int* block_offsets, unsigned int num_elems)
{
	extern __shared__ unsigned int sh_nums[];

	int local_index = threadIdx.x;
	int global_index = local_index + (blockDim.x * blockIdx.x);

	if(global_index >= num_elems)
	{
		return;
	}

	sh_nums[local_index] = nums[global_index];
	__syncthreads();

	for(int offset = 1; offset < blockDim.x; offset <<= 1)
	{
		int left = local_index - offset;
		int left_val = 0;
		if(left >= 0)
		{
			left_val = sh_nums[left];
		}
		__syncthreads();
		sh_nums[local_index] += left_val;
		__syncthreads();
	}

	if(local_index == 0)
	{
		c_nums[global_index] = 0;
	}
	else
	{
		c_nums[global_index] = sh_nums[local_index - 1];
	}

	if(local_index == blockDim.x - 1 || global_index == num_elems - 1)
	{
		block_offsets[blockIdx.x] = sh_nums[local_index];
	}

}

__global__
void scanInclusiveSum(unsigned int* nums, unsigned int* c_nums, unsigned int num_elems)
{
	extern __shared__ unsigned int sh_nums[];

    int local_index = threadIdx.x;
    int global_index = (blockDim.x * blockIdx.x) + local_index;

    if(global_index >= num_elems)
    {
        return;
    }

    sh_nums[local_index] = nums[global_index];
    __syncthreads();

    for(int offset = 1; offset < blockDim.x; offset <<= 1)
    {
        int left = local_index - offset;
        int left_val = 0;
        if(left >= 0)
        {
            left_val = sh_nums[left];
        }
        __syncthreads();
       	sh_nums[local_index] += left_val;
       
        __syncthreads();
    }

    c_nums[global_index] = sh_nums[local_index];
}

__global__ 
void addBlockOffsets(unsigned int* c_nums, unsigned int* block_offsets, unsigned int num_elems)
{
	int local_index = threadIdx.x;
    int global_index = (blockDim.x * blockIdx.x) + local_index;

    if(global_index >= num_elems)
    {
        return;
    }

    if(blockIdx.x == 0)
    {
    	return;
    }

    c_nums[global_index] += block_offsets[blockIdx.x - 1];
}

__global__
void scatterAddresses(unsigned int* vals_in, unsigned int* pos_in, unsigned int* vals_out, unsigned int* pos_out, unsigned int* flags, unsigned int* addresses, unsigned int offset, unsigned int num_elems)
{
	int local_index = threadIdx.x;
    int global_index = (blockDim.x * blockIdx.x) + local_index;

    if(global_index >= num_elems)
    {
        return;
    }

    if(flags[global_index] == 1)
    {
    	int addr = addresses[global_index] + offset;
    	vals_out[addr] = vals_in[global_index];
    	pos_out[addr] = pos_in[global_index];
    }
}

__global__
void setOrderFlag(unsigned int* vals, unsigned int* order_flag, unsigned int num_elems)
{
	int local_index = threadIdx.x;
    int global_index = (blockDim.x * blockIdx.x) + local_index;

    if(global_index >= num_elems)
    {
        return;
    }

    if(global_index == 0)
    {
    	return;
    }

    if(vals[global_index - 1] > vals[global_index])
    {
    	order_flag[0] = 1;
    }
}

bool isSorted(unsigned int* vals, unsigned int num_elems)
{
	unsigned int num_threads = 1024;
	unsigned int num_blocks = ceil(num_elems / (float) num_threads);

	unsigned int h_order_flag = 0;
	unsigned int* d_order_flag;
	hipMalloc(&d_order_flag, sizeof(unsigned int));
	hipMemset(d_order_flag, 0, sizeof(unsigned int));
	setOrderFlag<<<num_threads, num_blocks>>>(vals, d_order_flag, num_elems);
	hipMemcpy(&h_order_flag, &d_order_flag[0], sizeof(unsigned int), hipMemcpyDeviceToHost);
	return h_order_flag == 0;
}

void initRadixMemory(unsigned int num_elems)
{
	hipMalloc(&vals_buffer, sizeof(unsigned int) * num_elems);
	hipMalloc(&pos_buffer, sizeof(unsigned int) * num_elems);

	hipMalloc(&flags, sizeof(unsigned int) * num_elems);
	hipMalloc(&addresses, sizeof(unsigned int) * num_elems);
	hipMalloc(&block_offsets, sizeof(unsigned int) * ceil(num_elems / 1024.0f));
}

void radixSortFloat(unsigned int* vals, unsigned int* pos, unsigned int num_elems)
{
	unsigned int* vals_one = vals;
	unsigned int* vals_two = vals_buffer;
	unsigned int* pos_one = pos;
	unsigned int* pos_two = pos_buffer;

	unsigned int num_threads = 1024;
	unsigned int num_blocks = ceil(num_elems / (float) num_threads);

	convertFloats<<<num_blocks, num_threads>>>(vals_one, num_elems);

	for(unsigned int bit_pos = 0; bit_pos < 32; bit_pos++)
	{	
		if(isSorted(vals_one, num_elems))
		{
			//printf("in order!!! \n");
			break;
		}

		markFlags<<<num_blocks, num_threads>>>(vals_one, flags, false, bit_pos, num_elems);
		scanExclusiveSumWithBlockOffsets<<<num_blocks, num_threads, sizeof(unsigned int) * num_threads>>>(flags, addresses, block_offsets, num_elems);
		scanInclusiveSum<<<1, num_blocks, sizeof(int) * num_blocks>>>(block_offsets, block_offsets, num_blocks);
		addBlockOffsets<<<num_blocks, num_threads>>>(addresses, block_offsets, num_elems);
		scatterAddresses<<<num_blocks, num_threads>>>(vals_one, pos_one, vals_two, pos_two, flags, addresses, 0, num_elems);
		
		unsigned int offset = 0;
		hipMemcpy(&offset, &block_offsets[num_blocks - 1], sizeof(unsigned int), hipMemcpyDeviceToHost);
		//printf("number of 0's %d \n", offset);

		flipFlags<<<num_blocks, num_threads>>>(flags, num_elems);
		scanExclusiveSumWithBlockOffsets<<<num_blocks, num_threads, sizeof(unsigned int) * num_threads>>>(flags, addresses, block_offsets, num_elems);
		scanInclusiveSum<<<1, num_blocks, sizeof(int) * num_blocks>>>(block_offsets, block_offsets, num_blocks);
		addBlockOffsets<<<num_blocks, num_threads>>>(addresses, block_offsets, num_elems);
		scatterAddresses<<<num_blocks, num_threads>>>(vals_one, pos_one, vals_two, pos_two, flags, addresses, offset, num_elems);
	
		unsigned int* vals_temp = vals_one;
		vals_one = vals_two;
		vals_two = vals_temp;

		unsigned int* pos_temp = pos_one;
		pos_one = pos_two;
		pos_two = pos_temp;
	}
	invertFloats<<<num_blocks, num_threads>>>(vals_one, num_elems);
	hipMemcpy(vals, vals_one, sizeof(int) * num_elems, hipMemcpyDeviceToDevice);
	hipMemcpy(pos, pos_one, sizeof(int) * num_elems, hipMemcpyDeviceToDevice);
}

int main()
{
	srand((int) time(NULL));
	unsigned int num_elems = 1024 * 1024;

	initRadixMemory(num_elems);

	float* h_vals;
	h_vals = (float*) malloc(sizeof(float) * num_elems);

	unsigned int* h_pos;
	h_pos = (unsigned int*) malloc(sizeof(unsigned int) * num_elems);

	for(int i = 0; i < num_elems; i++)
	{
		h_vals[i] = (((float) rand() / RAND_MAX) * 1000) - 500;
		//h_vals[i] = (float) i;
		h_pos[i] = i;
	}

	unsigned int* d_vals;
	hipMalloc(&d_vals, sizeof(unsigned int) * num_elems);

	unsigned int* d_pos;
	hipMalloc(&d_pos, sizeof(unsigned int) * num_elems);

	hipMemcpy(d_vals, h_vals, sizeof(unsigned int) * num_elems, hipMemcpyHostToDevice);
	hipMemcpy(d_pos, h_pos, sizeof(unsigned int) * num_elems, hipMemcpyHostToDevice);

	radixSortFloat(d_vals, d_pos, num_elems);
	
	hipMemcpy(h_vals, d_vals, sizeof(unsigned int) * num_elems, hipMemcpyDeviceToHost);

	for(int i = 0; i < 100; i++)
	{
		printf("%f \n", h_vals[i]);
	}

	return 0;
}








